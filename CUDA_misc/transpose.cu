#include "hip/hip_runtime.h"
%%cu

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void transpose(int *d_A, int *d_B, int m, int n){
	int row = threadIdx.y;
	int col = threadIdx.x;

	d_B[col * m + row] = d_A[row * n + col];

}

int main(){
	int *h_A, *h_B, *d_A, *d_B;

	int m = 2, n = 3;
	int A[] = {1, 2, 3, 4, 5, 6};

	int size = m * n * sizeof(int);
	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);

	for(int i = 0; i < m*n; i++){
		h_A[i] = A[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(n, m, 1);
	transpose<<<grid, block>>>(d_A, d_B, m, n);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("original: \n");
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			printf("%d ", h_A[i * n + j]);
		}
		printf("\n");
	}
 
	printf("result: \n");
	for(int i = 0; i < n; i++){
		for(int j = 0; j < m; j++){
			printf("%d ", h_B[i * m + j]);
		}
		printf("\n");
	}

}