#include "hip/hip_runtime.h"
%%cu 

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define TW 2

__global__ void practice(int *d_M, int *d_N, int *d_P, int width){

	//elements brought in by collaboration stored in M and N
	__shared__ int M[TW][TW];
	__shared__ int N[TW][TW];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	//finding out row and column of the element in the output array that we want to work on
	int row = ty + by*TW;
	int col = tx + bx*TW;

	int product = 0;

	// no. of phases = width/TW
	// for loop to loop for no. of phases


	for(int i = 0; i < width/TW; i++){

		//collaborative loading of d_M and d_N tiles into shared memory
		// we are seeing which element the thread needs to bring in for each phase

		M[ty][tx] = d_M[row*width + TW*i + tx];
		N[ty][tx] = d_N[(i*TW +ty)*width + col];

		__syncthreads();

		//calculating partial sum
		for(int i = 0; i < TW; i++){
			product += M[ty][i]*N[i][tx];
		}
	}

	d_P[row*width + col] = product;
}


int main(){
	int *h_M, *h_N, *h_P, *d_N, *d_M, *d_P;

	int m = 4, n = 4;

	int M[] = {1, 2, 3, 4, 5, 6, 7, 8, 1, 2, 3, 4, 5, 6, 7, 8};
	int N[] = {1, 2, 3, 4, 5, 6, 7, 8, 1, 2, 3, 4, 5, 6, 7, 8};

	int size = m * n * sizeof(int);

	h_M = (int*)malloc(size);
	h_N = (int*)malloc(size);
	h_P = (int*)malloc(size);

	for(int i = 0; i < m*n ; i++){
		h_M[i] = M[i];
		h_N[i] = N[i];
	}

	hipMalloc((void**)&d_M, size);
	hipMalloc((void**)&d_N, size);
	hipMalloc((void**)&d_P, size);

	hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
	hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

	dim3 grid(m/TW, n/TW, 1);
	dim3 block(TW, TW, 1);
	practice<<<grid, block>>>(d_M, d_N, d_P, n);

	hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

	printf("result: \n");
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			printf("%d  ", h_P[i*n + j]);
		}
		printf("\n");
	}

}