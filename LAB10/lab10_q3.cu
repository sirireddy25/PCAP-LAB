#include "hip/hip_runtime.h"
%%cu 
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>


__global__ void q3(int *d_A, int *d_B, int m, int n){
	int row = threadIdx.y;
	int col = threadIdx.x;
	int idx = threadIdx.x + blockDim.x*threadIdx.y;

	if(row == 0|| col == 0 || row == m - 1 || col == n - 1){
		d_B[idx] = d_A[idx];
	}
	else{
		int num = d_A[idx];
		int i = 0, rem = 0, pow = 1;
		while(num != 0){
			rem += !(num % 2) * pow;
			num = num / 2;
			pow = pow * 10;
		}
		d_B[idx] = rem;
	}
}


int main(){
	int *h_A, *h_B, *d_A, *d_B;

	int m = 4, n = 4;

	int A[] = {1, 2, 3, 4, 6, 5, 8, 3, 2, 4, 10, 1, 9, 1, 2, 5};

	int size = m * n * sizeof(int);

	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);

	for(int i = 0; i < m*n; i++){
		h_A[i] = A[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(n, m, 1);

	q3<<<grid, block>>>(d_A, d_B, m, n);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("resultant matrix: \n");
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			printf("%d ", h_B[i*n +j]);
		}
		printf("\n");
	}
}
