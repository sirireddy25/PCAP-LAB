#include "hip/hip_runtime.h"
__global__ void convolution_1D_basic_kernel(float *N, float *P, int mask_width, int width){
	int i = blockDim.x*blockIdx.x + threadIdx.x;

	__shared__ NS[TW + mask_width - 1];

	int n = mask_width/2;

	halo_index_left = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
	if (threadIdx.x >= blockDim.x - n){
		NS[threadIdx.x - (blockDim.x - n)] = (halo_index_left < 0 ) ? 0 : N[halo_index_left];
	}

	NS[n + threadIdx.x] = N[i];

	int halo_index_right = (blockIdx.x + 1)*blockDim.x + threadIdx.x;
	if (threadIdx.x < n){
		NS[threadIdx.x + blockDim.x + n] = (halo_index_right >= width) ? 0 : N[halo_index_right];
	}

	__syncthreads();

	int pvalue;
	for(int j = 0; j < mask_width; j++){
		pvalue += NS[threadIdx.x + j] * M[j];
	}
	P[i] = pvalue;

}

grid size = width/TW
block size  = TW 