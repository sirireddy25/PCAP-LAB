#include "hip/hip_runtime.h"
//sparse matrix
%%cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sparse(int num_rows, int* data, int* col_index, int* row_ptr, int* x, int* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        int product = 0;
        int start = row_ptr[row];
        int stop = row_ptr[row + 1];
        for (int k = start; k < stop; k++) {
            product += data[k] * x[col_index[k]];
        }
        y[row] = product;
    }
}

int main() {
    int n = 4, m = 4;
    int* h_matrix = (int*)malloc(n * m * sizeof(int));
    int* h_x = (int*)malloc(m * sizeof(int));
    int non_zero_count = 0;
    int init_h_matrix[] = {3, 0, 0, 1, 0, 0, 0, 0, 0, 2, 4, 1, 1, 0, 0, 1};
    for (int i = 0; i < n * m; i++) {
        h_matrix[i] = init_h_matrix[i];
        non_zero_count += (h_matrix[i] != 0);
    }

    int init_h_x[] = {1, 2, 1, 2};
    for (int i = 0; i < m; i++) {
        h_x[i] = init_h_x[i];
    }

    int* h_data = (int*)malloc(non_zero_count * sizeof(int));
    int* h_col_index = (int*)malloc(non_zero_count * sizeof(int));
    int* h_row_ptr = (int*)calloc(n + 1, sizeof(int));
    int* h_y = (int*)calloc(n, sizeof(int));

    int k = 0, b = 0;
    for(int i = 0; i < n; i++){
        h_row_ptr[b++] = k;
        for(int j = 0; j < m; j++){
            if(h_matrix[i*m + j] != 0){
                h_col_index[k] = j;
                h_data[k] = h_matrix[i*m + j];
                k++;
            }
        }
    }
    h_row_ptr[b] = non_zero_count;

    int *d_data, *d_col_index, *d_row_ptr, *d_x, *d_y;
    hipMalloc((void**)&d_data, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_col_index, non_zero_count * sizeof(int));
    hipMalloc((void**)&d_row_ptr, (n + 1) * sizeof(int));
    hipMalloc((void**)&d_x, m * sizeof(int));
    hipMalloc((void**)&d_y, n * sizeof(int));
    
    hipMemcpy(d_data, h_data, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, h_col_index, non_zero_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, h_row_ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, m * sizeof(int), hipMemcpyHostToDevice);

    sparse<<<1, n>>>(n, d_data, d_col_index, d_row_ptr, d_x, d_y);
    hipMemcpy(h_y, d_y, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Y:");
    for (int i = 0; i < n; i++) {
        printf(" %d", h_y[i]);
    }
    printf("\n");
}

