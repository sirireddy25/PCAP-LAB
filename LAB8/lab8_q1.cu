#include "hip/hip_runtime.h"
%%cu
//WORD COUNT
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void wordcountkernel(char *d_str, char *d_key, int *d_index, int *d_count){
    int idx = threadIdx.x;
    int start;
    if(idx == 0){
        start = 0;
    }
    else{
        start = d_index[idx] + 1;
    }

    int end = d_index[idx + 1];

    int equal = 1;
    int j = 0;
    for(int i = start; i < end; i++){
        if(d_str[i] != d_key[j]){
            equal = 0;
            break;
        }
        j++;
    }
    if(equal == 1){
        atomicAdd(d_count, 1);
    }
}

int main(){
    char h_str[100], h_key[100];
    char *d_str, *d_key;
    int *d_count, *d_index; 
    int count, index[10];

    strcpy(h_str, "hello world bye world");
    int str_size = sizeof(char)*strlen(h_str);
    strcpy(h_key, "world");
    int key_size = sizeof(char)*strlen(h_key);

    int word_count = 0, k = 0;
    index[k++] = 0;
    for(int i = 0; i < strlen(h_str); i++){
        if(h_str[i] == ' '){
          word_count++;
          index[k++] = i;
        }
    }
    word_count++;
    index[k] = str_size; 

    count = 0;

    hipMalloc((void**)&d_str, str_size);
    hipMalloc((void**)&d_key, key_size);
    hipMalloc((void**)&d_index, (word_count + 1)*sizeof(int));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_str, h_str, str_size, hipMemcpyHostToDevice);
    hipMemcpy(d_key, h_key, key_size, hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, (word_count + 1)*sizeof(int), hipMemcpyHostToDevice);

    wordcountkernel<<<1, word_count>>>(d_str, d_key, d_index, d_count);

    hipError_t err = hipGetLastError();

    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("%s occurs %d times.", h_key, count);
}