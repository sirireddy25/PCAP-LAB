#include "hip/hip_runtime.h"
%%cu
//1D convolution
#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void conv(int *d_arr, int *d_mask, int *d_out, int width, int mask_width){
    int idx = threadIdx.x;
    int start_point = idx - (mask_width/2);
    int output = 0;
    for(int i = 0; i < mask_width; i++){
        if(start_point + i >= 0 && start_point + i < width){
            output = output + d_arr[start_point + i] * d_mask[i];
        }
    }
    d_out[idx] = output;
}

int main(){
    int *h_arr, *h_mask, *h_out, *d_out, *d_arr, *d_mask;
    
    int width = 6;
    int mask_width = 3;

    int arr_size = sizeof(int)*width;
    int mask_size = sizeof(int)*mask_width;

    h_arr = (int*)malloc(arr_size);
    h_out = (int*)malloc(arr_size);
    h_mask = (int*)malloc(mask_size);
    
    int mask[] = {1, 2, 1};
    int arr[] = {1, 2, 3, 4, 5, 6};

    for(int i = 0; i < width; i++){
        h_arr[i] = arr[i];
    }

    for(int i = 0; i < mask_width; i++){
        h_mask[i] = mask[i];
    }

    hipMalloc((void**)&d_arr, arr_size);
    hipMalloc((void**)&d_out, arr_size);
    hipMalloc((void**)&d_mask, mask_size);
    hipMemcpy(d_arr, h_arr, arr_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);

    conv<<<1, width>>>(d_arr, d_mask, d_out, width, mask_width);

    hipMemcpy(h_out, d_out, arr_size, hipMemcpyDeviceToHost);

    printf("resultant array: ");
    for(int i = 0; i < width; i++){
        printf("%d ", h_out[i]);
    }
}