#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#define TW 4

__device__ __constant__ int MW;
__device__ __constant__ int mask[5];

__global__ void conv(int *d_A, int *d_B, int width){
	
  	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	__shared__ int AS[TW];

	AS[threadIdx.x] = d_A[idx];

	__syncthreads();

	int this_tile_start_point = blockIdx.x*blockDim.x;
	int next_tile_start_point = (blockIdx.x + 1)*blockDim.x;

	int start_point = idx - (MW/2);

	int output = 0;
  	int j = 0;
	for(int i = 0; i < MW; i++){
		if( start_point + i >= 0 && start_point + i < width){
			if(start_point + i >= this_tile_start_point && start_point + i < next_tile_start_point){
				output += AS[j] * mask[i];
        		j++;
			}
			else{
				output += d_A[start_point + i]*mask[i];
			}
		}
	}

	d_B[idx] = output;
}

int main(){
	int *h_A,*h_B, *h_M, *d_A, *d_B;
	int mask_width, width;

	width = 16;
	mask_width = 5;

	int M[] = {1, 1, 2, 1, 1};
	int A[] = {10, 20, 30, 40, 50, 60, 70, 80, 90, 100, 11, 120, 130, 140, 150, 160};

	int size = width*sizeof(int);

	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);
	h_M = (int*)malloc(mask_width*sizeof(int));

	for(int i = 0; i < width; i++){
		h_A[i] = A[i];
	}

	for(int i = 0; i < mask_width; i++){
		h_M[i] = M[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(MW), &mask_width, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(mask), h_M, mask_width*sizeof(int), 0, hipMemcpyHostToDevice);

	dim3 grid(4, 1, 1);
	dim3 block(4, 1, 1);

	conv<<<grid, block>>>(d_A, d_B, width);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("result: \n");
	for(int i = 0; i < width; i++){
		printf("%d ", h_B[i]);
	}

}