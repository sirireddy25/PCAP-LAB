#include "hip/hip_runtime.h"
%%cu
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void q2a(char *d_A, int *d_B, char *d_C, int m, int n){
	int idx = threadIdx.x + blockDim.x * threadIdx.y;
	int offset = 0;

	for(int i = 0; i < idx; i++){
		offset += d_B[i];
	}

	int repeat = d_B[idx];

	for(int i  = 0; i < repeat; i++){
		d_C[offset++] = d_A[idx];
	}
}

int main(){
	char *h_A, *h_C, *d_A, *d_C;
	int *h_B, *d_B;

	int m = 2, n = 4;

	int sizeA = m * n * sizeof(char);
	int sizeB = m * n * sizeof(int);

	h_A = (char*)malloc(sizeA);
	h_B = (int*)malloc(sizeB);

	char A[] = {'p', 'C', 'a', 'P', 'e', 'X', 'a', 'M'};
	int B[] = {1, 2, 4, 3, 2, 4, 3, 2};
	int sum = 0;

	for(int i = 0; i < m*n; i++) {
        h_A[i] = A[i];
        h_B[i] = B[i];
        sum += B[i];
    }

    int sizeC = sizeof(char)*sum;
	h_C = (char*)malloc(sizeC);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 block(n, m, 1);

    q2a<<<grid, block>>>(d_A, d_B, d_C, m, n);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    printf("resultant string: %s", h_C);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}