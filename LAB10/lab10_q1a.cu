#include "hip/hip_runtime.h"
%%cu

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void q1a(int *d_A, int *d_B, int m, int n){
	int idx = threadIdx.x + blockDim.x*threadIdx.y;

	int row = threadIdx.y;
	int col = threadIdx.x;

	int sum = 0;

	for(int i = 0; i < n; i++){
		sum += d_A[row * n + i];
	}
	for(int i = 0; i < m; i++){
		sum += d_A[i * n + col];
	}

	d_B[idx] = sum;
}

int main(){
	int *h_A, *h_B, *d_A, *d_B;

	int m = 2, n = 3;

	int size = m * n * sizeof(int);

	int A[] = {1, 2, 3, 4, 5, 6};

	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);

	for(int i = 0; i < m*n; i++){
		h_A[i] = A[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(n, m, 1);

	q1a<<<grid, block>>>(d_A, d_B, m, n);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("result: \n");
	for(int i = 0; i < m*n; i++){
		printf("%d ", h_B[i]);
	}
}