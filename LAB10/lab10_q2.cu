#include "hip/hip_runtime.h"
%%cu

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

__glocal__ void q2(int *d_A, int *d_B, int n){
    int row = threadIdx.x;

    int ele;

    for(int i = 0; i < n; i++){
        ele = d_A[row * n + i];
        d_B[row * n + i] = powf(ele, row + 1);
    }

}

int main(){
    int *h_A, *h_B, *d_A, *d_B;

    int m = 3, n = 3;
    int A[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};

    int size = m * n * sizeof(int);

    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);

    for(int i = 0; i < m*n ; i++){
        h_A[i] = A[i];
    }

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    dim3 grid(1, 1, 1);
    dim3 block(m, 1, 1);
    q2<<<grid, block>>>(d_A, d_B, n);

    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < m; i++){
        for(int j = 0; j < n; j++){
            printf("%d  ", h_B[i * n + j]);
        }
        printf("\n");
    }
}