#include "hip/hip_runtime.h"
%%cu
//ADDING VECTORS
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(int *d_A, int *d_B, int *d_sum){
    int idx = threadIdx.x;
    d_sum[idx] = d_A[idx] + d_B[idx];
 }

int main(){
    int *h_A, *h_B, *d_B, *d_A, *h_sum, *d_sum;
    int n = 5;
    int size = sizeof(int)*n;
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_sum = (int*)malloc(size);
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_sum, size);

    int A[] = {1, 2, 3, 4, 5};
    int B[] = {1, 2, 3, 4, 5};
    for(int i = 0; i < 5; i++){
        h_A[i] = A[i];
        h_B[i] = B[i];
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    vector_add<<<1, n>>>(d_A, d_B, d_sum);

    hipMemcpy(h_sum, d_sum, size, hipMemcpyDeviceToHost);

    printf("sum :\n");
    for(int i = 0; i < 5; i++){
        printf("%d  ", h_sum[i]);
    }

}
