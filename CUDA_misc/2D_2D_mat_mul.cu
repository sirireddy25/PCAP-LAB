#include "hip/hip_runtime.h"
//Write a CUDA program to perform matrix multiplication using 2D grid and 2D block
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

_global_ void q1(int *d_A, int *d_B, int *d_C, int ha, int wa, int hb, int wb){
    int r = blockIdx.y*blockDim.y + threadIdx.y;
    int c = blockIdx.x*blockDim.x + threadIdx.x;
    int sum = 0;

    if( r < ha && c < wb){
        for(int k = 0; k < wa; k++){
            sum += d_A[r*wa + k] * d_B[k*wb + c];
        }
        d_C[r*wb + c] = sum;
    }
}

int main(){
    int *h_A, *h_B, *d_A, *d_B, *h_C, *d_C;
    int ha, wa, hb, wb;
    int sizeA, sizeB, sizeC;

    printf("enter dimensions of matrix A: \n");
    scanf("%d %d", &ha, &wa);

    printf("enter dimensions of matrix B: \n");
    scanf("%d %d", &hb, &wb);


    sizeA = ha*wa*sizeof(int);
    sizeB = hb*wb*sizeof(int);
    sizeC = ha*wb*sizeof(int);

    h_A = (int*)malloc(sizeA);
    h_B = (int*)malloc(sizeB);
    h_C = (int*)malloc(sizeC);

    hipMalloc((void**)&d_A, sizeA);    
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    printf("enter elements of matrix A: \n");
    for(int i = 0; i < ha; i++){
        for(int j = 0; j < wa; j++){
            scanf("%d", &h_A[i*wa + j]);
        }
    }

    printf("enter elements of matrix B: \n");
    for(int i = 0; i < hb; i++){
        for(int j = 0; j < wb; j++){
            scanf("%d", &h_B[i*wb + j]);
        }
    }

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 block(16, 16, 1);
    dim3 grid(ceil(wb/2.0), ceil(ha/2.0), 1);
    q1<<<grid, block>>>(d_A, d_B, d_C, ha, wa, hb, wb);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    printf("result: \n");
    for(int i = 0; i < ha; i++){
        for(int j = 0; j < wb; j++){
            printf("%d  ", h_C[i*wb +j]);
        }
        printf("\n");
    }

}