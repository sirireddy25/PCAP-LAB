#include "hip/hip_runtime.h"
%%cu
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void q2a(int *d_A, int *d_B, int n){
	int row = threadIdx.y;
	int col = threadIdx.x;
	int idx = threadIdx.x + blockDim.x*threadIdx.y;

	int num, fact = 1, sum = 0;

	if(row == col){
		d_B[idx] = 0;
	}
	else if(col > row){
		num = d_A[idx];
		for(int i = 1; i <= num; i++){
			fact = fact * i;
		}
		d_B[idx] = fact;
	}
	else{
		num = d_A[idx];
		while(num != 0){
			sum += num % 10;
			num = num / 10;
		}
		d_B[idx] = sum;
	}
}

int main(){
	int *h_A, *h_B, *d_A, *d_B;

	int n = 3;
	int A[] = {1, 2, 3, 14, 5, 6, 17, 18, 9};

	int size = n * n * sizeof(int);

	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);

	for(int i = 0; i < n*n; i++){
		h_A[i] = A[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(3, 3, 1);

	q2a<<<grid, block>>>(d_A, d_B, n);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("resultant matrix: \n");
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			printf("%d ", h_B[i*n +j]);
		}
		printf("\n");
	}


}