#include "hip/hip_runtime.h"
%%cu
//1D parallel convolution using constant memory
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__device__ __constant__ int MW;
__device__ __constant__ int mask[3];

__global__ void conv(int* d_A, int* d_B, int width){
	int idx = threadIdx.x;

	int start_point = idx - (MW/2);

	int output = 0;
	for(int i = 0; i < MW; i++){
		if(start_point + i >= 0 && start_point + i < width ){
			output += d_A[start_point + i] * mask[i];
		}
	}

	d_B[idx] = output;
}

int main(){
	int *h_A, *h_B, *h_M, *d_A, *d_B;
	int width = 7;
	int mask_width = 3;
	int size = width * sizeof(int);

	int A[] = {1, 2, 3, 4, 5, 6, 7};
	int M[] = {1, 2, 1};

	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);
	h_M = (int*)malloc(mask_width*sizeof(int));

	for(int i = 0; i < width; i++){
		h_A[i] = A[i];
	}
 
	for(int i = 0; i < mask_width; i++){
	     h_M[i] = M[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(mask ), h_M, sizeof(int)*mask_width, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(MW ), &mask_width, sizeof(int), 0, hipMemcpyHostToDevice);
 
	dim3 grid(1, 1, 1);
	dim3 block(width, 1, 1);
	conv<<<grid, block>>>(d_A, d_B, width);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("result: \n");
	for(int i = 0; i < width; i++){
		printf("%d ", h_B[i]);
	}

}