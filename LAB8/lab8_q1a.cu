#include "hip/hip_runtime.h"
%%cu 

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void q1a(char *d_A, char *d_B, int *d_index){
	int idx = threadIdx.x;
	int start, end, len;
	if(idx == 0){
		start = 0;
	}
	else{
		start = d_index[idx] + 1;
	}

	end = d_index[idx + 1] - 1;
	int j = start;
	for(int i = end; i >= start; i--){
		d_B[j++] = d_A[i];
	}
	d_B[j] = ' ';
}

int main(){
	char *d_A, *d_B;
  	int *d_index;
	char h_A[100], h_B[100];

	strcpy(h_A, "hello world bye world");

	int size = strlen(h_A)*sizeof(char);
	int index[10];
	int wordcount = 0, k = 0;

	index[k++] = 0;
	for(int i = 0; i < strlen(h_A); i++){
		if(h_A[i] == ' '){
			wordcount++;
			index[k++] = i;
		}
	}
	wordcount++;
	index[k] = strlen(h_A);

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_index, (wordcount + 1)*sizeof(int));


	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_index, index, (wordcount + 1)*sizeof(int) , hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(wordcount, 1, 1);
	q1a<<<grid, block>>>(d_A, d_B, d_index);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

	printf("result: %s\n", h_B);

}