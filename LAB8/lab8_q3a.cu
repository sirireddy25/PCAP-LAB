#include "hip/hip_runtime.h"
%%cu

#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<string.h>

__global__ void q3a(char *d_sin, char*d_T){
	int idx = threadIdx.x;

	int letter = d_sin[idx];

	int start = 0;

	if (idx == 0){
		start = 0;
	}
	else{
		for(int i = 1; i <= idx; i++){
			start += i;
		}
	}

	for(int i = 0; i < idx + 1; i++){
		d_T[start] = letter;
    start += 1;
	}


}

int main(){
	char str[] = "Hai";
	int len = strlen(str);

	int Tlen = 0;
	for(int i = 1; i <= len; i++){
		Tlen += i;
	}

	char sin[len], T[Tlen];
	char *d_sin, *d_T;

	strcpy(sin, str);

	int sin_size = len*sizeof(char);
	int T_size = Tlen*sizeof(char);

	hipMalloc((void**)&d_sin, sin_size);
	hipMalloc((void**)&d_T, T_size);

	hipMemcpy(d_sin, sin, sin_size, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(len, 1, 1);
	q3a<<<grid, block>>>(d_sin, d_T);

	hipMemcpy(T, d_T, T_size, hipMemcpyDeviceToHost);

	printf("result: %s\n", T);
}