#include"hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>

__global__ void row(int *A, int *B, int *C, int ha, int wa, int hb, int wb){
	int r = threadIdx.x;
	for(int c = 0; c < wb; c++){
		int sum = 0;
		for(int k = 0; k < wa; k++){
			sum += A[r * wa + k] * B[k * wb + c];
		}
		C[r * wb + c] = sum;
	}
}

__global__ void col(int *A, int *B, int *C, int ha, int wa, int hb, int wb){
	int c = threadIdx.x;
	for(int r = 0; r < ha; r++){
		int sum = 0;
		for(int k = 0; k < wa; k++){
			sum += A[r * wa + k] * B[k * wb + c];
		}
		C[r * wb + c] = sum;
	}
}

__global__ void ele(int *A, int *B, int *C, int ha, int wa, int hb, int wb){
	int r = threadIdx.y;
	int c = threadIdx.x;
	int sum = 0;
	for(int k = 0; k < wa; k++){
		sum += A[r * wa + k] * B[k * wb + c];
	}
	C[r * wb + c] = sum;
}

int main(){
	int *A, *B, *C;
	int *dA, *dB, *dC;
	int ha, wa, hb, wb;
	printf("enter dimensions of matrix A: \n");
	scanf("%d %d", &ha, &wa);
	printf("enter dimensions of matrix B: \n");
	scanf("%d %d", &hb, &wb);

	if(wa != hb){
		printf("invalid dimensions!\n");
		exit(1);
	}

	int sizeA, sizeB, sizeC;

	sizeA = ha*wa*sizeof(int);
	sizeB = hb*wb*sizeof(int);
	sizeC = ha*wb*sizeof(int);

	A = (int*)malloc(sizeA);
    B = (int*)malloc(sizeB);
    C = (int*)malloc(sizeC);

    printf("enter elements of matrix A:\n");
    for(int i = 0; i < ha; i++){
    	for(int j = 0; j < wa; j++){
    		scanf("%d", &A[i*wa + j]);
    	}
    }

    printf("enter elements of matrix B:\n");
    for(int i = 0; i < hb; i++){
    	for(int j = 0; j < wb; j++){
    		scanf("%d", &B[i*wb + j]);
    	}
    }


    hipMalloc((void**)&dA, sizeA);
    hipMalloc((void**)&dB, sizeB);
    hipMalloc((void**)&dC, sizeC);
    hipMemcpy(dA, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeB, hipMemcpyHostToDevice);

    row<<<1, ha>>>(dA, dB, dC, ha, wa, hb, wb);
    hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost);

    printf("\neach row computed by one thread: \n");
    for(int i = 0; i < ha; i++){
    	for(int j = 0; j < wb; j++){
    		printf("%d ", C[i*wb + j]);
    	}
    	printf("\n");
    }


    col<<<1, wb>>>(dA, dB, dC, ha, wa, hb, wb);
    hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost);

    printf("\neach column computed by one thread: \n");
    for(int i = 0; i < ha; i++){
    	for(int j = 0; j < wb; j++){
    		printf("%d ", C[i*wb + j]);
    	}
    	printf("\n");
    }

    dim3 grid(1, 1, 1);
    dim3 block(wb, ha, 1);
    ele<<<grid, block>>>(dA, dB, dC, ha, wa, hb, wb);
    hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost);

    printf("\neach element computed by one thread: \n");
    for(int i = 0; i < ha; i++){
    	for(int j = 0; j < wb; j++){
    		printf("%d ", C[i*wb + j]);
    	}
    	printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    return 0;

}