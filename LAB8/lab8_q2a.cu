#include "hip/hip_runtime.h"
%%cu

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void q2a(char *d_sin, char *d_sout, int len, int n){
	int idx = threadIdx.x;

	char letter = d_sin[idx];
	int offset = idx;

	for(int i = 0; i < n; i++){
		d_sout[offset] = letter;
		offset += len;
	}

}

int main(){
	int n = 3;
	char str[] = "hello";
	int len = strlen(str);
	char sin[len], sout[len*n];
	char *d_sin, *d_sout;

	strcpy(sin, "Hello");

	int size_sin = len*sizeof(char);
	int size_sout = n*len*sizeof(char);

	hipMalloc((void**)&d_sin, size_sin);
	hipMalloc((void**)&d_sout, size_sout);

	hipMemcpy(d_sin, sin, size_sin, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(len, 1, 1);
	q2a<<<grid, block>>>(d_sin , d_sout, len, n);

	hipMemcpy(sout, d_sout, size_sout, hipMemcpyDeviceToHost);

	printf("result: %s\n", sout);

}