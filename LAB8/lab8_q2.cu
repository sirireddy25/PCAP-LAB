#include "hip/hip_runtime.h"
%%cu
//S = PCAP RS = PCAPPCAPCP
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

__global__ void repeat(char *d_S, char *d_RS, int *d_index, int S_len){
    int idx = threadIdx.x;
    int r = S_len - idx;
    int start = d_index[idx];
    for(int i = 0; i < r; i++){
      d_RS[start++] = d_S[i];
    }
}

int main(){
    char h_S[5], h_RS[20];
    char *d_S, *d_RS;
    int S_len, RS_len, S_size, RS_size ;

    strcpy(h_S, "PCAP");
    S_len = strlen(h_S);
    int index[S_len], *d_index;

    RS_len = 0;
    int j = 0;
    for(int i = S_len; i >=0; i--){
        index[j++] = RS_len;
        RS_len += i;
    }

    S_size = S_len*sizeof(char);
    RS_size = RS_len*sizeof(char);

    hipMalloc((void**)&d_S, S_size);
    hipMalloc((void**)&d_RS, RS_size);
    hipMalloc((void**)&d_index, S_len*sizeof(int));

    hipMemcpy(d_S, h_S, S_size, hipMemcpyHostToDevice);
    hipMemcpy(d_index, index, S_len*sizeof(int), hipMemcpyHostToDevice);

    repeat<<<1, S_len>>>(d_S, d_RS, d_index, S_len);
    hipError_t err = hipGetLastError();

    printf("resultant string: ");
    for(int i = 0; i < RS_len; i++){
        printf("%c", h_RS[i]);
    }

}