#include "hip/hip_runtime.h"
%%cu
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void q1a(int *d_A, int *d_B, int m, int n){
	int row = threadIdx.y;
	int col = threadIdx.x;
	int idx = threadIdx.x + blockDim.x*threadIdx.y;

	int sum = 0;
	if(d_A[idx] %2 == 0){
		for(int k = 0; k < n; k++){
			sum += d_A[row*n + k];
		}
	}
	else{
		for(int k = 0; k < n; k++){
			sum += d_A[k * n + col];
		}
	}

	d_B[idx] = sum;
}


int main(){
	int *h_A, *h_B, *d_A, *d_B;
	int m = 2, n = 3;

	int A[] = {1, 2, 3, 4, 5, 6};

	int size = m*n*sizeof(int);

	h_A = (int*)malloc(size);
	h_B = (int*)malloc(size);

	for(int i = 0; i < m*n; i++){
		h_A[i] = A[i];
	}

	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);


	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	dim3 grid(1, 1, 1);
	dim3 block(n, m, 1);

	q1a<<<grid, block>>>(d_A, d_B, m, n);

	hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);


	printf("resultant matrix: \n");
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			printf("%d  ", h_B[i*n + j]);
		}
		printf("\n");
	}

}
